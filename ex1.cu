#include "hip/hip_runtime.h"
#include "ex1.h"

__device__ void prefix_sum(int arr[], int arr_size) 
{
    int tid = TILE_WIDTH * threadIdx.y + threadIdx.x;
    int increment;
    for (int stride = 1; stride < arr_size; stride *= 2) 
    {
        if (tid >= stride) 
        // I think we should add limit condition between tid
        // and array size for example if tid - stride> arr_size 
            increment = arr[tid - stride];
        __syncthreads(); 
        if (tid >= stride) 
            arr[tid] += increment;
        __syncthreads();
    }
    return;
}

__device__ void convert_image_to_tiles(uchar *tile, uchar image_in[IMG_WIDTH][IMG_HEIGHT])
{   
    // calculate pixel index in parallel threads
    int in_pixel_index_x = blockIdx.x * TILE_WIDTH + threadIdx.x; 
    int in_pixel_index_y = (blockIdx.y * TILE_WIDTH + threadIdx.y) * IMG_WIDTH;
    int tile_pixel_index = threadIdx.y * TILE_DIM + threadIdx.x;

    // assign input pixel to tile pixel value
    tile[tile_pixel_index] = images_in[in_pixel_index_x][in_pixel_index_y];
}

__device__ void calculate_maps(int *cdf, uchar maps[TILES_COUNT][TILES_COUNT][N_BINS])
{
    double div_result = 0.0;
    
    if (threadIdx.x < N_BINS)
    {
        div_result = (float)cdf[threadIdx.x] * NORMALIZATION_FACTOR;
        maps[blockIdx.x][blockIdx.y][threadIdx.x] = (uchar)div_result;
    }        
}

__device__ void create_histogram(int *histograms, uchar *tile)
{
    //We can accelerate this compute - https://classroom.udacity.com/courses/cs344/lessons/5605891d-c8bf-4e0d-8fed-a47920df5979/concepts/b42e8f5a-9145-450e-8c18-f23e091d33ef
    uchar pixel_value = 0;

    // initialize histogram
    if(threadIdx.x < N_BINS)
    {
	    histograms[threadIdx.x] = 0;
    }
    __syncthreads();

    pixel_value = tile[threadIdx.x + TILE_WIDTH * threadIdx.y];
    atomicAdd(&(histograms[pixel_value]), 1);
}

/**
 * Perform interpolation on a single image
 *
 * @param maps 3D array ([TILES_COUNT][TILES_COUNT][256]) of    
 *             the tiles’ maps, in global memory.
 * @param in_img single input image, in global memory.
 * @param out_img single output buffer, in global memory.
 */
__device__ void interpolate_device(uchar* maps ,uchar *in_img, uchar* out_img);

/**
 * @brief takes an image given in all_in, and return the processed image in all_out 
 * 
 * @param all_in single input image, in global memory.
 * @param all_out single output image, in global memory.
 * @param maps 3D array ([TILES_COUNT][TILES_COUNT][256]) of    
 *             the tiles’ maps, in global memory.
 * @return __global__ 
 */
__global__ void process_image_kernel(uchar *all_in, uchar *all_out, uchar *maps) 
{
    //every block allocates the shared-mem for itself
    //we need only two array for this calculation
    __shared__ int cdf[N_BINS];
    __shared__ uchar tile[TILE_WIDTH * TILE_WIDTH];

    convert_image_to_tiles(tile, all_in)

    create_histogram(cdf, tile)

    prefix_sum(cdf, N_BINS);

    calculate_maps(cdf, maps)

    interpolate_device(all_in, all_out, maps);

    return; 
}

/* Task serial context struct with necessary CPU / GPU pointers to process a single image */
struct task_serial_context
 {
    // TODO define task serial memory buffers
    uchar image_in[IMG_WIDTH][IMG_HEIGHT];
    uchar image_out[IMG_WIDTH][IMG_HEIGHT];
    uchar maps[TILES_COUNT][TILES_COUNT][N_BINS];
};

/* Allocate GPU memory for a single input image and a single output image.
 * 
 * Returns: allocated and initialized task_serial_context. */
struct task_serial_context *task_serial_init()
{
    auto context = new task_serial_context;

    //TODO: allocate GPU memory for a single input image, a single output image, and maps
    CUDA_CHECK( hipHostAlloc(context->image_in, IMG_HEIGHT * IMG_WIDTH, 0) );
    CUDA_CHECK( hipHostAlloc(context->image_out, IMG_HEIGHT * IMG_WIDTH, 0) );
    CUDA_CHECK( hipHostAlloc(context->maps, TILES_COUNT * TILES_COUNT * N_BINS, 0) );

    return context;
}

/* Process all the images in the given host array and return the output in the
 * provided output host array */
void task_serial_process(struct task_serial_context *context, uchar *images_in, uchar *images_out)
{
    //TODO: in a for loop:
    //   1. copy the relevant image from images_in to the GPU memory you allocated
    //   2. invoke GPU kernel on this image
    //   3. copy output from GPU memory to relevant location in images_out_gpu_serial
    
    
    dim3 BLOCK_SIZE(N_BLOCKS_X, N_BLOCKS_Y);
    dim3 GRID_SIZE(TILE_WIDTH, TILE_WIDTH);

    int image_index = 0;

    for (; image_index < N_IMAGES ; ++image_index)
    {
         //   1. copy the relevant image from images_in to the GPU memory you allocated
        CUDA_CHECK( hipMemcpy(context->image_in, &images_in[image_index * IMG_WIDTH * IMG_HEIGHT], IMG_WIDTH * IMG_HEIGHT * sizeof(uchar), hipMemcpyHostToDevice) );
        
        //   2. invoke GPU kernel on this image
        process_image_kernel<<<BLOCK_SIZE, GRID_SIZE>>>(&(context->image_in), &(context->image_out), context->maps);

        //   3. copy output from GPU memory to relevant location in images_out_gpu_serial
        CUDA_CHECK( hipMemcpy(context->image_out, &images_out[image_index * IMG_WIDTH * IMG_HEIGHT], IMG_WIDTH * IMG_HEIGHT * sizeof(uchar), hipMemcpyDeviceToDevice) );
    }

}
/* Release allocated resources for the task-serial implementation. */
void task_serial_free(struct task_serial_context *context)
{
    //TODO: free resources allocated in task_serial_init
    free(context->image_in);
    free(context->image_out));
    free(context->maps));
    free(context);
}

/* Bulk GPU context struct with necessary CPU / GPU pointers to process all the images */
struct gpu_bulk_context {
    // TODO define bulk-GPU memory buffers
};

/* Allocate GPU memory for all the input images, output images, and maps.
 * 
 * Returns: allocated and initialized gpu_bulk_context. */
struct gpu_bulk_context *gpu_bulk_init()
{
    auto context = new gpu_bulk_context;

    //TODO: allocate GPU memory for all the input images, output images, and maps

    return context;
}

/* Process all the images in the given host array and return the output in the
 * provided output host array */
void gpu_bulk_process(struct gpu_bulk_context *context, uchar *images_in, uchar *images_out)
{
    //TODO: copy all input images from images_in to the GPU memory you allocated
    //TODO: invoke a kernel with N_IMAGES threadblocks, each working on a different image
    //TODO: copy output images from GPU memory to images_out
}

/* Release allocated resources for the bulk GPU implementation. */
void gpu_bulk_free(struct gpu_bulk_context *context)
{
    //TODO: free resources allocated in gpu_bulk_init

    free(context);
}
